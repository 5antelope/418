#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

extern float toBW(int bytes, float sec);

// Kernel definition
// Device code
__global__ void
saxpy_kernel(int N, float alpha, float* x, float* y, float* result) {

    // compute overall index from position of thread in current block,
    // and given the block we are in
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N) {
       result[index] = alpha * x[index] + y[index];
    }
}

// Host code
void
saxpyCuda(int N, float alpha, float* xarray, float* yarray, float* resultarray) {

    int totalBytes = sizeof(float) * 3 * N;

    // compute number of blocks and threads per block
    const int threadsPerBlock = 512;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    float* device_x;
    float* device_y;
    float* device_result;

    //
    // TODO allocate device memory buffers on the GPU using hipMalloc
    //
    int size = sizeof(float) * N;
    hipMalloc((void **)&device_x, size);
    hipMalloc((void **)&device_y, size);
    hipMalloc((void **)&device_result, size);



    //
    // TODO copy input arrays to the GPU using hipMemcpy
    //
    hipMemcpy(device_x, xarray, size, hipMemcpyHostToDevice);
    hipMemcpy(device_y, yarray, size, hipMemcpyHostToDevice);

    // start timing after allocation of device memory
    double startTime = CycleTimer::currentSeconds();
    // run kernel
    saxpy_kernel<<<blocks, threadsPerBlock>>>(N, alpha, device_x, device_y, device_result);
    hipDeviceSynchronize();
    // end timing after result has been copied back into host memory
    double endTime = CycleTimer::currentSeconds();


    //
    // TODO copy result from GPU using hipMemcpy
    //
    hipError_t err = hipMemcpy(resultarray, device_result, size, hipMemcpyDeviceToHost);
    #ifdef DEBUG
    fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", err, hipGetErrorString(err));
    #endif



    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    printf("CUDA: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));

    // TODO free memory buffers on the GPU
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_result);
}

void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}


//#ifdef DEBUG
//    double startTime = CycleTimer::currentSeconds();
//#endif
//#ifdef DEBUG
//    double endTime = CycleTimer::currentSeconds();
//    dbg_printf("Thread %d completes! Time:[%.3f] ms\n", args->threadId, 1000*(endTime-startTime));
//#endif